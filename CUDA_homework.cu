
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 12

#define THREADS_X 3
#define THREADS_Y 4

#define A(i,j) A[i*N+j]
#define B(i,j) B[i*N+j]
#define C(i,j) C[i*N+j]

__global__ void index(int *A, int *B, int *C)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	C(i,j) = A(i,j) + B(i,j);
}

int main()
{
   int A[N*N], B[N*N], C[N*N], *A_d, *B_d, *C_d;
   int i, j;

	dim3 dimBlock(THREADS_X, THREADS_Y);
	dim3 dimGrid(N/THREADS_X, N/THREADS_Y);

	hipMalloc((void **)&A_d, sizeof(int)*N*N);
	hipMalloc((void **)&B_d, sizeof(int)*N*N);
	hipMalloc((void **)&C_d, sizeof(int)*N*N);

	// 초기화
   for (i=0; i<N; i++){
		for(j=0; j<N; j++){
			A(i,j) = i+j;
	      B(i,j) = i*j;
		}
	}

	hipMemcpy(A_d, A, sizeof(int)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, sizeof(int)*N*N, hipMemcpyHostToDevice);

	index <<< dimGrid, dimBlock >>> (A_d, B_d, C_d);

	hipMemcpy(C, C_d, sizeof(int)*N*N, hipMemcpyDeviceToHost);

	for(i=0; i<N; i++){
		for(j=0; j<N; j++){
			printf("%d ", C(i,j));
		}
		printf("\n");
	}

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
}
