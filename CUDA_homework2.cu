
#include <hip/hip_runtime.h>
#include<stdio.h>

#define N 160
#define THREADS 16

__global__ void sum(float *A, float *B, float *result_d)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	result_d[i] = A[i] * B[i];

	__shared__ float sresult[THREADS];

	sresult[threadIdx.x] = result_d[i];

	for(unsigned int s = blockDim.x/2; s>0; s>>=1)
	{
		if(threadIdx.x < s){
			sresult[threadIdx.x] += sresult[threadIdx.x+s];
			__syncthreads();
		}
	}
	if(threadIdx.x == 0){
		result_d[blockIdx.x] = sresult[threadIdx.x];
	}
}

int main()
{
   float A[N], B[N], tresult, *result;
	float *A_d, *B_d, *result_d;
	int i;
	
	dim3 dimBlock(THREADS);
	dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x);

	for(i=0; i<N; i++){
		A[i] = i * 2;
		B[i] = N - i;
	}

	hipMalloc((void **) &A_d, sizeof(float)*N);
	hipMalloc((void **) &B_d, sizeof(float)*N);
	hipMalloc((void **) &result_d, sizeof(float)*dimGrid.x);

	hipMemcpy(A_d, A, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, sizeof(float)*N, hipMemcpyHostToDevice);

	sum<<<dimGrid, dimBlock>>>(A_d, B_d, result_d);

	result = (float*)malloc(sizeof(float)*dimGrid.x);
	hipMemcpy(result, result_d, sizeof(float)*dimGrid.x, hipMemcpyDeviceToHost);

	tresult = 0.0;
	for(i=0; i<dimGrid.x; i++){
		tresult += result[i];
	}

	printf("GPU dotprod : %f\n", tresult);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(result_d);
	free(result);
}

